#include "hip/hip_runtime.h"
/*
	Based on Tanguy Pruvot's repo
	Provos Alexis - 2016
	Optimized for nvidia pascal by sp (2018)
*/

#include "cuda_helper_alexis.h"
#include "cuda_vectors_alexis.h"
#include "cuda_x11_aes_alexis.cuh"
#include "cubehash/cubehash512.cuh"
#include "shavite/shavite512.cuh"

#define TPB 1024
#define CUBEHASH_SHAVITE_TPB 128

/***************************************************/
// GPU Hash Function
__global__
void x11_cubehash512_gpu_hash_64(uint32_t threads, uint64_t *g_hash){

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads){
		x11_cubehash512_gpu_hash_64_unroll_10r((uint32_t*)&g_hash[8 * thread]);
	}
}

__global__
//__launch_bounds__(384, 2)
__launch_bounds__(CUBEHASH_SHAVITE_TPB, 3)
void x11_cubehashShavite512_gpu_hash_64(uint32_t threads, uint32_t *g_hash){

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    __shared__ uint32_t sharedMemory[4][256];
    aes_gpu_init128(sharedMemory);

    uint32_t *const hash = &g_hash[thread << 4];

    x11_cubehash512_gpu_hash_64(hash);
    __syncthreads();
    shavite512(sharedMemory, hash);
}


__host__
void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash){

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + TPB-1)/TPB);
    dim3 block(TPB);

    x11_cubehash512_gpu_hash_64<<<grid, block>>>(threads, (uint64_t*)d_hash);
}

__host__
void x11_cubehash_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash)
{

	dim3 grid((threads + CUBEHASH_SHAVITE_TPB - 1) / CUBEHASH_SHAVITE_TPB);
	dim3 block(CUBEHASH_SHAVITE_TPB);

	x11_cubehashShavite512_gpu_hash_64 << <grid, block >> > (threads, d_hash);
}
