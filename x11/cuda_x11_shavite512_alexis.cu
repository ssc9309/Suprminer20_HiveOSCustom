#include "hip/hip_runtime.h"
/*
	Based on Tanguy Pruvot's repo
	Provos Alexis - 2016
*/
#include "cuda_helper_alexis.h"
#include "cuda_vectors_alexis.h"

#define INTENSIVE_GMF
#include "cuda_x11_aes_alexis.cuh"
#include "shavite/shavite512.cuh"

#define TPB 128

// GPU Hash
//__global__ __launch_bounds__(TPB,8) /* 5820 */
__global__ __launch_bounds__(TPB,7) /* 5900 */
//__global__ __launch_bounds__(TPB, 6) /* 5775 */
void x11_shavite512_gpu_hash_64_alexis(const uint32_t threads, uint32_t *g_hash) {
    __shared__ uint32_t sharedMemory[4][256];
	aes_gpu_init128(sharedMemory);

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {
      uint32_t *hash = &g_hash[thread<<4];
      __syncthreads();
      shavite512(sharedMemory, hash);
    }
}

__host__
void x11_shavite512_cpu_hash_64_alexis(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	// note: 128 threads minimum are required to init the shared memory array
	x11_shavite512_gpu_hash_64_alexis<<<grid, block>>>(threads, d_hash);
}
